#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <string>
#include <fstream> 
#include <hip/hip_runtime.h>
#include ""

// CUDA kernel to count words in parallel
__global__ void countWordsGPU(const char* words, int* count, int numWords) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int partialCount = 0;

    for (int i = tid; i < numWords; i += stride) {
        if (words[i] == ' ' || words[i] == '\n') {
            partialCount++;
        }
    }

    atomicAdd(count, partialCount);
}

int main() {
    std::string filename;
    std::cout << "Enter the filename: ";
    std::cin >> filename;

    // Read the document from the file
    std::ifstream inputFile(filename);
    if (!inputFile) {
        std::cerr << "Error opening file." << std::endl;
        return 1;
    }

    std::string document;
    std::string line;
    while (std::getline(inputFile, line)) {
        document += line;
        document += "\n";
    }
    inputFile.close();

    // Convert the document to a char array to transfer to GPU
    char* wordsGPU;
    hipMalloc(&wordsGPU, document.size() + 1);
    hipMemcpy(wordsGPU, document.c_str(), document.size() + 1, hipMemcpyHostToDevice);

    // Count the number of words on the GPU
    int* wordCountGPU;
    hipMalloc(&wordCountGPU, sizeof(int));
    hipMemset(wordCountGPU, 0, sizeof(int));

    // GPU configuration
    int threadsPerBlock = 256;
    int numBlocks = (document.size() + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the GPU kernel
    countWordsGPU << <numBlocks, threadsPerBlock >> > (wordsGPU, wordCountGPU, document.size());

    // Transfer the result back to CPU
    int wordCount;
    hipMemcpy(&wordCount, wordCountGPU, sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    // Free memory on GPU
    hipFree(wordsGPU);
    hipFree(wordCountGPU);

    std::cout << "Total number of words in the document: " << wordCount << std::endl;

    return 0;
}
